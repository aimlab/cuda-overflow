
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
using namespace std;
//for different kernel
#define BUF_LEN 16
#define N  2
typedef unsigned long(*pFdummy)(void);

__device__ __noinline__ unsigned long dummy1()
{
	return 0x1111111111111111;
}
__device__ __noinline__ unsigned long dummy2()
{
	return 0x2222222222222222;
}
__device__ __noinline__ unsigned long dummy3()
{
	return 0x3333333333333333;
}
__device__ __noinline__ unsigned long dummy4()
{
	return 0x4444444444444444;
}
__device__ __noinline__ unsigned long dummy5()
{
	return 0x5555555555555555;
}
__device__ __noinline__ unsigned long dummy6()
{
	return 0x6666666666666666;
}
__device__ __noinline__ unsigned long dummy7()
{
	return 0x7777777777777777;
}
__device__ __noinline__ unsigned long dummy8()
{
	return 0x8888888888888888;
}
__device__ __noinline__ unsigned long dummy9()
{
	return 0x9999999999999999;
}

__device__  unsigned long __noinline__ unsafe(unsigned int *input,int len)
{
	unsigned int buf[BUF_LEN];
	pFdummy fp[8];
	fp[0]=dummy1;
	fp[1]=dummy2;
	fp[2]=dummy3;
	fp[3]=dummy4;
	fp[4]=dummy5;
	fp[5]=dummy6;
	fp[6]=dummy7;
	fp[7]=dummy8;
	unsigned int hash=5381;
	//copy input to buf
	//printf("%x %x %x");
	printf("%p\n",dummy9);
	if(blockDim.x==2)
	for(int i=0;i<len;i++)
	{
		buf[i]=input[i];
		//printf("%x",input[i]);
	}


	//djb2
	for(int i=0;i<BUF_LEN;i++)
	{
		hash=((hash<<5)+hash)+buf[i];
		printf("%d\n", hash%8 );
	}
	return (unsigned long) (fp[hash%8])();
}

__global__ void test_kernel(unsigned long *hashes,unsigned int *input,int len,int admin)
{
	unsigned long my_hash;
	//int m;
	//m=*len;
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	printf("blockdim: %d,idx: %d, len: %d\n",blockDim.x, idx, len);


	if(admin)
	{	my_hash=dummy9();

}
	else
		my_hash=unsafe(input,len);
	hashes[idx]=my_hash;
}
__global__ void test_kernel2(unsigned long *hashes2,unsigned int *input,int len,int admin)
{
	unsigned long my_hash;
	//int m;
	//m=*len;
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	printf("blockdim: %d,idx: %d, len: %d\n",blockDim.x, idx, len);


	if(admin)
		my_hash=dummy9();

	else
		my_hash=unsafe(input,len);
	hashes2[idx]=my_hash;
}

static void checkCudaErrorAux(const char*file,unsigned line,const char*statement,hipError_t error)
{
	if(error==hipSuccess)
		return;
	cout<<statement<<"returned:"<<hipGetErrorString(error)<<"at file:"<<file<<"line:"<<line<<endl;
	exit(1);
}
#define CUDA_CHECK_RETURN(value) checkCudaErrorAux(__FILE__,__LINE__,#value,value)

int main()
{
	unsigned int input[100];
	int len=27,admin=0;
	unsigned long hashes[N];
	unsigned long hashes2[N];
	unsigned long *dev_hashes;
	unsigned long *dev_hashes2;
	unsigned int *dev_input;
	unsigned int m=0;

	m=0x24;
	//m=0x450;

	//cout<<"start!"<<endl;
		for(int i=0;i<len;i++)
			input[i]=m;

	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_hashes,N*sizeof(unsigned long)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_hashes2,N*sizeof(unsigned long)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_input,100*sizeof(unsigned int)));
	CUDA_CHECK_RETURN(hipMemcpy(dev_input,input,100*sizeof(unsigned int),hipMemcpyHostToDevice));
	test_kernel<<<1,N>>>(dev_hashes,dev_input,len,admin);
	test_kernel2<<<1,1>>>(dev_hashes2,dev_input,len,admin);
	CUDA_CHECK_RETURN(hipMemcpy(hashes,dev_hashes,N*sizeof(unsigned long),hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(hashes2,dev_hashes2,N*sizeof(unsigned long),hipMemcpyDeviceToHost));

	for(int i=0;i<N;i++)
	{
		printf("hash %lx\n", hashes[i]);
	}
	for(int i=0;i<1;i++)
	{
		printf("hash2 %lx\n", hashes2[i]);
	}

	CUDA_CHECK_RETURN(hipFree(dev_input));
	CUDA_CHECK_RETURN(hipFree(dev_hashes));
	
	return 0;
}

