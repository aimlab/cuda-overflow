
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
#include<stdio.h>
using namespace std;
//two thread in same block and considering global memory 
//overflow fo global memory cause all thread corrupted 
#define BUF_LEN 16
#define N 32
typedef unsigned long(*pFdummy)(void);

__device__ __noinline__ unsigned long dummy1()
{
	return 0x1111111111111111;
}
__device__ __noinline__ unsigned long dummy2()
{
	return 0x2222222222222222;
}
__device__ __noinline__ unsigned long dummy3()
{
	return 0x3333333333333333;
}
__device__ __noinline__ unsigned long dummy4()
{
	return 0x4444444444444444;
}
__device__ __noinline__ unsigned long dummy5()
{
	return 0x5555555555555555;
}
__device__ __noinline__ unsigned long dummy6()
{
	return 0x6666666666666666;
}
__device__ __noinline__ unsigned long dummy7()
{
	return 0x7777777777777777;
}
__device__ __noinline__ unsigned long dummy8()
{
	return 0x8888888888888888;
}
__device__ __noinline__ unsigned long dummy9()
{
	return 0x9999999999999999;
}
__device__ int overf[100];
__device__  unsigned long __noinline__ unsafe(int len)
{
	int buf[BUF_LEN];
	pFdummy fp[8];
	fp[0]=dummy1;
	fp[1]=dummy2;
	fp[2]=dummy3;
	fp[3]=dummy4;
	fp[4]=dummy5;
	fp[5]=dummy6;
	fp[6]=dummy7;
	fp[7]=dummy8;

	unsigned int hash=5381;
	//copy input to buf
	//printf("%x %x %x");
	printf("%d\n",len);
	//printf("%p\n",&buf[21]);
	//printf("%p\n",&fp[5]);
	printf("%p\n",dummy9);
	for(int i=0;i<len;i++)
	{
		buf[i]=overf[i];
	}
//buf[-6]=input[0];//shared mermoy中

	//djb2
	for(int i=0;i<BUF_LEN;i++)
	{
		hash=((hash<<5)+hash)+buf[i];
		printf("%d\n", hash%8 );
	}
	return (unsigned long) (fp[hash%8])();
}

__global__ void test_kernel(unsigned long *hashes,unsigned int *input,int len,int admin)
{
	unsigned long my_hash;
	//int m;
	//m=*len;
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	printf("idx: %d, len: %d\n", idx, len);
	if(threadIdx.x==0)
	for(int i=0;i<len;i++)
		overf[i]=input[i];

	if(admin)
	{	my_hash=dummy9();
		//my_hash=dummy8();
//	printf("%p\n",&idx);
	
}
	else
		{
		if(idx==0)
			my_hash=unsafe(len);
		else
			my_hash=unsafe(len);
		}
	hashes[idx]=my_hash;
}

static void checkCudaErrorAux(const char*file,unsigned line,const char*statement,hipError_t error)
{
	if(error==hipSuccess)
		return;
	cout<<statement<<"returned:"<<hipGetErrorString(error)<<"at file:"<<file<<"line:"<<line<<endl;
	exit(1);
}
#define CUDA_CHECK_RETURN(value) checkCudaErrorAux(__FILE__,__LINE__,#value,value)

int main()
{
	unsigned int input[100];
	int len=27,admin=0;
	unsigned long hashes[N];
	unsigned long *dev_hashes;
	unsigned int *dev_input;
	unsigned int m=0;
	m=0x590;
	//m=0x450;

	//cout<<"start!"<<endl;
		for(int i=0;i<len;i++)
			input[i]=m;


	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_hashes,N*sizeof(unsigned long)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_input,100*sizeof(unsigned int)));
	CUDA_CHECK_RETURN(hipMemcpy(dev_input,input,100*sizeof(unsigned int),hipMemcpyHostToDevice));

	test_kernel<<<1,N>>>(dev_hashes,dev_input,len,admin);
	CUDA_CHECK_RETURN(hipMemcpy(hashes,dev_hashes,N*sizeof(unsigned long),hipMemcpyDeviceToHost));

	for(int i=0;i<N;i++)
	{
		printf("%lx\n", hashes[i]);
	}

	CUDA_CHECK_RETURN(hipFree(dev_input));
	CUDA_CHECK_RETURN(hipFree(dev_hashes));
	
	return 0;
}
