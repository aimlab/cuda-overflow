/*
 ============================================================================
 Name        : structoverflow.cu
 Author      : 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
using namespace std;
#define BUF_LEN 6
static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
__device__ __noinline__ void normal()
{
	printf("normal!\n");
}
__device__ __noinline__ void secret()
{
	 printf("Hello Admin!\n");
}


struct unsafe
{
	unsigned long buf[BUF_LEN];
    void (*normal)();
};
__device__ __noinline__ void init(struct unsafe *data)
{
 data->normal=normal;
}
__global__ void test_kernel(unsigned long *input,int len,int admin)
{
	struct unsafe cu;
	init(&cu);
	for(int i=0;i<len;i++)
			cu.buf[i]=input[i];
	cu.normal();
	secret();
printf("%p",secret);

}
int main(void)
{
	unsigned long input[10];
	unsigned long *dev_input;
	int len=6;
	int admin=0;
	for(int i=0;i<10;i++)
	{
		input[i]=0xb2140;//this is secret（） address
	}
	CUDA_CHECK_RETURN(hipMalloc((void**)&dev_input,10*sizeof(unsigned long)));
	CUDA_CHECK_RETURN(hipMemcpy(dev_input,input,10*sizeof(unsigned long),hipMemcpyHostToDevice));
	test_kernel<<<1,1>>>(dev_input,len,admin);
	hipFree(dev_input);
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

